#include "hip/hip_runtime.h"
#include "GImageTransform.h"
#include "Common.h"
#include <hip/hip_runtime.h>
#include <cstdio>
__global__ void copy(float * from ,float * to,int size,int width,int height, int offset,int color){
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int x = id%(width)-offset;
	if(x<0) x=2-(-x-1)%3;
	if(x>=(width-2*offset)) x = width-2*offset-3+(x%3);
	
	int y = id/width-offset/3;
	if(y<0) y=0;
	if(y>=height-2*offset/3)y=height-2*offset/3-1;
	if(id < size)to[id] = (float)from[y*(width-2*offset) +x];
}

void GImageTransform::stretch(int val){
	int newH = img_->height()+2*val;
	int newW = img_->width()+2*val;
	int newSize = newH*newW*3;
	float * newMatrix;
	hipMalloc(&newMatrix,newSize*sizeof(float));

	int threadsPerBlock = MAX_THREADS;
	int blocksPerGrid = (newSize+threadsPerBlock-1) / threadsPerBlock;
	copy<<<blocksPerGrid,threadsPerBlock>>>(img_->data(),newMatrix,newSize,newW*3,newH,val*3,0);
	/*
	float *buf;
	buf = new float[newSize]();
	hipMemcpy(buf,newMatrix,newSize*sizeof(float),hipMemcpyDeviceToHost);
	for(int i = 0 ; i < newSize ; i++){
		if(i%(newW*3)==0)printf("\n");
		if(i%3==0)printf(" ");
		printf("%3.0f ",buf[i]);
	}
	printf("\n");
	printf("\n");
	
	hipMemcpy(buf,img_->data_,img_->size_*sizeof(float),hipMemcpyDeviceToHost);
	for(int i = 0 ; i < img_->size_ ; i++){
		if(i%(img_->width_*3)==0)printf("\n");
		if(i%3==0)printf(" ");
		printf("%3.0f ",buf[i]);
	}
	printf("\n");
	printf("\n");
	*/
	hipFree(img_->data_);
	delete img_->buf_;
	img_->buf_ = new uchar[newSize]();//kocham Wiewiórka
	img_->data_ = newMatrix;
	img_->width_ = newW;
	img_->height_ = newH;
	img_->size_ = newSize;
	 
}

GImageTransform::GImageTransform(GImage * img):
	img_(img)
{
	
}

GImageTransform::~GImageTransform()
{
	
}

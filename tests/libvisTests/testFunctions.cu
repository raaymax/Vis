#include "hip/hip_runtime.h"
#include "testFunctions.h"


__global__ void KrandF( float * f, int size){
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < size) f[id] = id%256;
}

void randomize(float * gpu , int size){
	int threads = 1024;
	int blocks = (size+threads-1)/threads;
	KrandF<<<blocks,threads>>>(gpu,size);
}

void randomizeU(uchar * cpu , int size){
	for(int i = 0 ; i < size ; i ++){
		cpu[i] = i%256;
	}
}

void printU(const uchar * data , int size){
	for(int i = 0 ; i < size ; i ++){
		printf("%d ",data[i]);
	}
	printf("\n");
}

void printPix(const uchar *data , int width , int height){
	for(int y = 0 ; y < height ; y++){
		for(int x = 0 ; x < width*3 ; x+=3){
			for(int i = 0 ; i < 3 ; i++){
				printf("%2d ",data[y*width*3+x+i]);
			}
			printf(" ");
		}
		printf("\n");
	}
	printf("\n");
	
}

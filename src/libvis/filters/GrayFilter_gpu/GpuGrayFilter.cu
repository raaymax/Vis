#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "GpuGrayFilter.h"
#include <exceptions/CudaException.h>

__global__ void process(uchar * data , int size){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	i*=3;
	data[i]=data[i+1]=data[i+2]=(data[i]+data[i+1]+data[i+2])/3;
}


// exported function, usable in the C++ part
extern "C" void runProcess(uchar* data , int size) {
	int nDevices;
	hipGetDeviceCount(&nDevices);
	if(nDevices < 1) throw CudaException(CudaException::NODEVICE);
	
	int n = size/3;
	size_t size_ = size * sizeof(uchar);
	uchar* data_;
	

	hipMalloc((void**)&data_, size_);
	hipMemcpy(data_, data, size_, hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocksPerGrid = (n+threadsPerBlock-1) / threadsPerBlock;
	
	process<<<blocksPerGrid, threadsPerBlock>>>(data_, size_);

	hipMemcpy(data, data_, size, hipMemcpyDeviceToHost);
	
	hipFree(data_);

}



GpuGrayFilter::GpuGrayFilter()
{

}

void GpuGrayFilter::process(Image & img){
	runProcess(img.data() , img.size());
}

GpuGrayFilter::~GpuGrayFilter()
{

}


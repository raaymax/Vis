#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <plotter/Plotter.h>
#include <QApplication>
#include "Matrix.h"
#include <boost/date_time/posix_time/posix_time.hpp>


__global__ 
void convolution(Matrix m , Matrix f, Matrix o){
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	int idy = blockIdx.y*blockDim.y+threadIdx.y;
	__shared__ float sm[5][5];
	if(idx < 5 && idy < 5) sm[idx][idy] = f.data[idy*5+idx];
	__syncthreads();
	for(int x = -2 ; x < 3 ; x++){
		for(int y = -2 ; y < 3 ; y++){
			if((y+idy) > 0 && (y+idy) < m.height && (x+idx) > 0 && (x+idx) < m.width)
				o.data[idy*m.width+idx] += m.data[(y+idy)*m.width+x+idx]*sm[2+y][2+x];
		}
	}
}

void function1(){
	Matrix i = create(32,32);
	randomize(i);
	printf("%s\n",hipGetErrorString(hipGetLastError()));
	
	Matrix o = create(32,32);
	Matrix f = create(5,5,CPU);
	f.data[2*f.width + 2] = 1;
	f.data[0*f.width + 2] = 0.5;
	f.data[4*f.width + 2] = 0.5;
	Matrix cf = copy(f,hipMemcpyHostToDevice);
	convolution<<<dim3(8,8),dim3(32,32)>>>(i,cf,o);
	printf("%s\n",hipGetErrorString(hipGetLastError()));
	Matrix c = copy(o,hipMemcpyDeviceToHost);
	printf("%s\n",hipGetErrorString(hipGetLastError()));
	for(int y = 0 ; y < c.height; y++){
		for(int x = 0 ; x < c.width ; x++){
			printf("%2.0f ",c.data[y*c.width + x]);
		}
		printf("\n");
	}
	printf("\n");
	
}

Matrix conv(const Matrix & A ,const Matrix &f){
	printf("%s",__FUNCTION__);
	Matrix n;
	if(A.type == CPU){
		printf("OK");
		if(f.type == GPU){
			printf("WTF2");
			throw;
		}
		n = create(A.width,A.height,CPU);
		for(int y = f.height/2 ; y < A.height-f.height/2 ;y++ ){
			for(int x = f.width/2 ; x < A.width-f.width/2 ;x++ ){
				for(int a = -(f.height/2) ; a < f.height/2 ;a++ ){
					for(int b = -(f.width/2) ; b < f.width/2 ;b++ ){
						if((y+a > 0) && (y+a < A.height) && (x+b > 0) && (x+b < A.width)){
							n.data[y*n.width+x] += A.data[(y+a)*A.width+(x+b)]*f.data[a*f.width+b];
						
							printf(".");
						}else{
							printf("%d %d",y+a,x+b);
						}
						printf("b");
					}
					printf("a");
				}
				printf("x");
			}
			printf("y");
		}
		
	}else{
		printf("WTF");
	}
	return n;
}

boost::posix_time::time_duration process(void (*func)()){
	boost::posix_time::ptime mst1 = boost::posix_time::microsec_clock::local_time();
	func();
	boost::posix_time::ptime mst2 = boost::posix_time::microsec_clock::local_time();
	return (mst2 - mst1);
}



int main(int argc , char ** argv){
//	QApplication app(argc,argv);
//	Plotter plotter;
//	Plot * p = plotter.createPlot(Qt::red);
//	plotter.show();
	for(int i = 0 ; i < 1 ; i++){
//		p->add(process(function1).total_milliseconds());
	}
	Matrix a = create(256,256,CPU);
	randomize(a);
	Matrix f = create(5,5,CPU);
	f.data[2*f.width+2]=1.0;
	Matrix c = conv(a,f);
//	print(c);

//	return app.exec();
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include "Matrix.h"


__global__ void rand(Matrix x){
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	int idy = blockDim.y*blockIdx.y + threadIdx.y;
	int id = idy * x.width + idx;
	x.data[id] = id%256;
}

Matrix create(uint w, uint h , MatrixType type){
	Matrix m;
	m.width = w;
	m.height = h;
	m.type = type;
	m.size = w*h*sizeof(float);
	if(type == CPU){
		m.data = (float*)malloc(m.size);
		memset(m.data,0,m.size);
	}else{
		hipMalloc(&m.data,m.size);
		hipMemset(m.data,0,m.size );
	}
	return m;
}

void randomize(Matrix & m){
	if(m.type == GPU){
		int dx=32,dy=32;
		while(m.height < dy) dy/=2;
		while(m.width < dx) dx/=2;
		dim3 t(dx,dy);
		dim3 b(m.width/32 , m.height/32);
		rand<<<b,t>>>(m);
	}else{
		for(int i = 0 ; i < m.width* m.height ; i++){
			m.data[i] = i%256;
		}
	}
}

Matrix copy(const Matrix & m, hipMemcpyKind kind){
	Matrix nm;
	if(kind == hipMemcpyDeviceToHost || kind == hipMemcpyHostToHost)
		nm = create(m.width,m.height,CPU);
	else
		nm = create(m.width,m.height,GPU);
	
	hipMemcpy(nm.data,m.data,m.size, kind);
	return nm;
}


void print(const Matrix &c){
	for(int y = 0 ; y < c.height; y++){
		for(int x = 0 ; x < c.width ; x++){
			printf("%2.0f ",c.data[y*c.width + x]);
		}
		printf("\n");
	}
	printf("\n");
}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "GImage.h"
#include <cstdio>
#include "Common.h"

__global__ void Kzero(float * f, int size){
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < size) f[id] = 0.0;
}

__global__ void KcopyFU(float * f, uchar * u , int size){
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < size) u[id] = (uchar)f[id];
}
__global__ void KcopyUF( uchar * u, float * f, int size){
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < size) f[id] = (float)u[id];
}



GImage::GImage():
	width_(0),
	height_(0),
	size_(0),
	data_(NULL)
{
	buf_ = NULL;
}

GImage::GImage(uint w, uint h):
	width_(w),
	height_(h),
	size_(w*h*3)
{
	int threads = MAX_THREADS;
	int blocks = (size_+threads-1)/threads;
	hipMalloc(&data_,size_*sizeof(float));
	Kzero<<<blocks,threads>>>(data_,size_);
	
	buf_ = new uchar[size_]();
}
GImage::GImage(const GImage& other){
	width_ = other.width();
	height_ = other.height();
	size_ = other.size();
	hipMalloc(&data_,size_*sizeof(float));
	hipMemcpy(data_,other.data(),size_*sizeof(float),hipMemcpyDeviceToDevice);
	buf_ = new uchar[size_]();
}

GImage::GImage(const IImage& other){
	width_ = other.width();
	height_ = other.height();
	size_ = other.size();
	buf_ = new uchar[size_]();
	hipMalloc(&data_,size_*sizeof(float));
	
	uchar * buf;
	hipMalloc(&buf,size_);
	hipMemcpy(buf, other.constData() , size_ , hipMemcpyHostToDevice);
	int threads = MAX_THREADS;
	int blocks = (size_+threads-1)/threads;
	KcopyUF<<<blocks,threads>>>(buf,data_,size_);
	hipFree(buf);
}

uint GImage::width()const{
	return width_;
}
uint GImage::height()const{
	return height_;
}
uint GImage::size() const{
	return size_;
}
float * GImage::data(){
	return data_;
}
const float * GImage::data() const{
	return data_;
}
const uchar * GImage::constData() const{
	if(data_==NULL){
		return NULL;
	}else{
		int threads = MAX_THREADS;
		int blocks = (size_+threads-1)/threads;
		uchar * buf;
		hipMalloc(&buf,size_);
		KcopyFU<<<blocks,threads>>>(data_,buf,size_);
		hipMemcpy(buf_,buf,size_,hipMemcpyDeviceToHost);
		hipFree(buf);
		return buf_;
	}
}


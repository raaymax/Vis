#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <string.h>
#include "Matrix.h"
#include <hip/hip_runtime_api.h>
#define DEBUG

#ifdef DEBUG
#define debug() printf("%s:%d %s#%s\n",__FILE__,__LINE__,__FUNCTION__,hipGetErrorString(hipGetLastError()))
#else
#define debug() 
#endif

__global__ void cuda_randomize(Matrix x){
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	int idy = blockDim.y*blockIdx.y + threadIdx.y;
	int id = idy * x.width + idx;
	if(id < x.count)
		x.data[id] = id%256;
}

Matrix matrix_create(uint w, uint h , MatrixType type){
	Matrix m;
	m.width = w;
	m.height = h;
	m.type = type;
	m.size = w*h*sizeof(float);
	m.count = w*h;
	if(type == CPU){
		m.data = (float*)malloc(m.size);
		memset(m.data,0,m.size);
	}else{
		hipMalloc(&m.data,m.size);
		debug();
		hipMemset(m.data,0,m.size );
		debug();
	}

	return m;
}

void matrix_destroy(Matrix & A){
	if(A.type == CPU){
		free(A.data);
	}else{
		hipFree(A.data);
		debug();
	}
	
	A.data = NULL;
	A.size = 0;
	A.data = 0;
	A.type = NONE;
}

void matrix_randomize(Matrix & m){
	assert(m.type != NONE);
	if(m.type == GPU){
		int dx=32,dy=32;
		while(m.height < dy) dy/=2;
		while(m.width < dx) dx/=2;
		dim3 t(dx,dy);
		dim3 b(m.width/32 , m.height/32);
		cuda_randomize<<<b,t>>>(m);
		debug();
	}else{
		for(int i = 0 ; i < m.width* m.height ; i++){
			m.data[i] = i%256;
		}
	}
}

Matrix matrix_copy(const Matrix & m, cudaMCKind kind){
	assert(m.type != NONE);
	Matrix nm;
	if(((int)kind)%2==1)
		nm = matrix_create(m.width,m.height,GPU);
	else
		nm = matrix_create(m.width,m.height,CPU);
	
	hipMemcpy(nm.data,m.data,m.size, static_cast<hipMemcpyKind>(kind));
	debug();
	return nm;
}

Matrix matrix_copy(const Matrix & m, MatrixType type){
	if(m.type == type){
		if(type==CPU){
			return matrix_copy(m,H2H);
		}else{
			return matrix_copy(m,D2D);
		}
	}else{
		if(type==CPU){
			return matrix_copy(m,D2H);
		}else{
			return matrix_copy(m,H2D);
		}
	}
}


void matrix_print(const Matrix &c){
	assert(c.type == CPU);
	for(int y = 0 ; y < c.height; y++){
		for(int x = 0 ; x < c.width ; x++){
			printf("%2.0f ",c.data[y*c.width + x]);
		}
		printf("\n");
	}
	printf("\n");
	debug();
}

__global__ 
void cuda_convolution(Matrix m , Matrix f, Matrix o){
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	int idy = blockIdx.y*blockDim.y+threadIdx.y;
	extern __shared__ float sm[];
	if(threadIdx.x < f.width && threadIdx.y < f.height) sm[threadIdx.y*f.width+threadIdx.x ] = f.data[threadIdx.y*f.width+threadIdx.x ];
	__syncthreads();
	int my = (f.height/2);
	int mx = (f.width/2);
	if(idy < m.height && idx < m.width){
		float r = 0.0;
		for(int y = -my ; y <= my ; y++){
			for(int x = -mx ; x <= mx; x++){
				if((y+idy) >= 0 && (y+idy) < m.height && (x+idx) >= 0 && (x+idx) < m.width)
					 r += m.data[(y+idy)*m.width+ x+idx]*sm[ (y+my)*f.width+x+mx ];
				
			}
		}
		o.data[idy*o.width+idx] = r;
	}
	
}

__global__
void cuda_normalize(Matrix m){
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	int idy = blockIdx.y*blockDim.y+threadIdx.y;
	if(idx < m.width && idy < m.height){
		m.data[idy*m.width+idx] = round(m.data[idy*m.width+idx]);
		if(m.data[idy*m.width+idx] < 0)
			m.data[idy*m.width+idx] = 0;
		else if(m.data[idy*m.width+idx] > 255)
			m.data[idy*m.width+idx] = 255;
	}
}



float matrix_convolution(int x, int y,const Matrix & A ,const Matrix &f){
	assert(A.type == CPU && f.type == CPU);
	float r = 0.0;
	for(int a = -(f.height/2) ; a <= (int)(f.height/2) ;a++ ){
		for(int b = -(f.width/2) ; b <= (int)(f.width/2) ;b++ ){
			if((y+a >= 0) && (y+a < A.height) && (x+b >= 0) && (x+b < A.width)){
				r += A.data[(y+a)*A.width+(x+b)]*f.data[(a+(f.height/2))*f.width+(b+(f.width/2))];
			}
		}
	}
	return r;
}


Matrix matrix_convolution(const Matrix & A ,const Matrix &f){
	assert(A.type != NONE && f.type != NONE);
	Matrix n;
	if(A.type == CPU){
		assert(f.type != GPU);
		
		float sum = 0;
		for(int i = 0 ; i < f.size/4 ; i++){
			sum += f.data[i];
		}
		n = matrix_create(A.width,A.height,CPU);
		for(int y = 0 ; y < A.height ;y++ ){
			for(int x = 0 ; x < A.width ;x++ ){
				n.data[y*n.width+x]= matrix_convolution(x, y,A ,f);
	/*			n.data[y*n.width+x] = round(n.data[y*n.width+x]);
				if(A.data[y*A.width+x] < 0)
					A.data[y*A.width+x]= 0;
				else if(A.data[y*A.width+x] > 255)
					A.data[y*A.width+x] = 255;
	*/
			}
		}
		
		
	}else{
		assert(f.type != CPU);
		n = matrix_create(A.width,A.height,GPU);
		//	convolution<<<dim3(8,8),dim3(32,32)>>>(A,f,n);
		int dx=32,dy=32;
		while(A.height < dy) dy/=2;
		while(A.width < dx) dx/=2;
		dim3 t(dx,dy);
		dim3 b((A.width+dx)/dx , (A.height+dy)/dy);
	//	printf("%d %d %d %d\n",dx,dy,(A.width+dx)/dx,(A.height+dy)/dy);
		cuda_convolution<<<b,t,f.size>>>(A,f,n);
	//	normalize<<<dim3(8,8),dim3(32,32)>>>(n);
		debug();
	}
	return n;
}

__global__ 
void cuda_divide(Matrix A , float val){
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	int idy = blockIdx.y*blockDim.y+threadIdx.y;
	if(idx < A.width && idy < A.height)
	A.data[idy*A.width+idx] /= val;
}

void matrix_divide(Matrix & A , float val){
	assert(A.type != NONE);
	if(A.type == CPU){
		for(int y = 0 ; y < A.height ;y++ ){
			for(int x = 0 ; x < A.width ;x++ ){
				A.data[y*A.width+x] /= val; 
			}
		}
	}else{
		int dx=32,dy=32;
		while(A.height < dy) dy/=2;
		while(A.width < dx) dx/=2;
		dim3 t(dx,dy);
		dim3 b((A.width+dx)/dx , (A.height+dy)/dy);
		cuda_divide<<<b,t>>>(A,val);
		debug();
	}
}

void matrix_normalize(Matrix & A){
	assert(A.type != NONE);
	if(A.type == CPU){
		for(int y = 0 ; y < A.height ;y++ ){
			for(int x = 0 ; x < A.width ;x++ ){
				A.data[y*A.width+x] = round(A.data[y*A.width+x]);
				if(A.data[y*A.width+x] < 0)
					A.data[y*A.width+x]= 0;
				else if(A.data[y*A.width+x] > 255)
					A.data[y*A.width+x] = 255;
			}
		}
	}else{
		int dx=32,dy=32;
		while(A.height < dy) dy/=2;
		while(A.width < dx) dx/=2;
		dim3 t(dx,dy);
		dim3 b(A.width/dx , A.height/dy);
		cuda_normalize<<<b,t>>>(A);
		debug();
	}
}

float matrix_sum(const Matrix &A){
	assert(A.type != NONE);
	float sum_=0.0;
	if(A.type == CPU){
		for(int i = 0 ; i< A.count ;i++ ){
			sum_ += A.data[i];
		}
	}else{
		//TODO: sumowanie elementow macierzy
	}
	return sum_;
}

__host__ __device__ 
float get(const Matrix & A , int x , int y){
	if(x < 0 )x = 0;
	if(y < 0 ) y=0;
	if(x >= A.width) x = A.width-1;
	if(y >= A.height) y = A.height-1;
	return A.data[y*A.width+x];
}
__global__
void cuda_resize(Matrix A , Matrix o){
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	int idy = blockIdx.y*blockDim.y+threadIdx.y;
	int nx = ((int)A.width - (int)o.width)/2;
	int ny = ((int)A.height - (int)o.height)/2;
	if(idx < o.width && idy < o.height)
	o.data[idy*o.width+idx] = get(A,idx+nx,idy+ny);
}

Matrix matrix_resize(const Matrix &A , uint width , uint height){
	assert(A.type != NONE);
	Matrix n;
	if(A.type == CPU){
		int nx = ((int)A.width - (int)width)/2;
		int ny = ((int)A.height - (int)height)/2;
		
		n = matrix_create(width,height,CPU);
		for(int y = 0 ; y < height ; y++){
			for(int x = 0 ; x < width ; x++){
			//	printf("%d %d\n",x+nx,y+ny);
				n.data[y*width+x] = get(A,x+nx,y+ny);
			}
		}
		
	}else{
		n = matrix_create(width,height,GPU);
		int dx=32,dy=32;
		while(n.height < dy) dy/=2;
		while(n.width < dx) dx/=2;
		dim3 t(dx,dy);
		dim3 b((n.width+dx)/dx , (n.height+dy)/dy);
		cuda_resize<<<b,t>>>(A,n);
		debug();
	}
	return n;
}

bool matrix_compare(const Matrix &A ,const Matrix &B){
	assert(A.type == CPU && A.type == B.type);
	if(A.width != B.width) return false; 
	if(A.height != B.height) return false; 
	if(A.size != B.size) return false; 
	if(memcmp(A.data,B.data,A.size)!=0)return false;
	return true;
}




